/*
 * 1_check_dimension.cu
 *
 * grid.x 2 grid.y 1 grid.z 1
 * block.x 3 block.y 1 block.z 1
 *
 * threadIdx:(0,0,0) blockIdx:(0,0,0) blockDim:(3,1,1)  gridDim(2,1,1)
 * threadIdx:(1,0,0) blockIdx:(0,0,0) blockDim:(3,1,1)  gridDim(2,1,1)
 * threadIdx:(2,0,0) blockIdx:(0,0,0) blockDim:(3,1,1)  gridDim(2,1,1)
 * threadIdx:(0,0,0) blockIdx:(1,0,0) blockDim:(3,1,1)  gridDim(2,1,1)
 * threadIdx:(1,0,0) blockIdx:(1,0,0) blockDim:(3,1,1)  gridDim(2,1,1)
 * threadIdx:(2,0,0) blockIdx:(1,0,0) blockDim:(3,1,1)  gridDim(2,1,1)
 */
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void checkIndex(void) {
  printf("threadIdx:(%d,%d,%d) blockIdx:(%d,%d,%d) blockDim:(%d,%d,%d) "
         "gridDim(%d,%d,%d)\n",
         threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y,
         blockIdx.z, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y,
         gridDim.z);
}

int main(int argc, char **argv) {
  int nElem = 6;
  dim3 block(3);
  dim3 grid((nElem + block.x - 1) / block.x);
  printf("grid.x %d grid.y %d grid.z %d\n", grid.x, grid.y, grid.z);
  printf("block.x %d block.y %d block.z %d\n", block.x, block.y, block.z);
  checkIndex<<<grid, block>>>();
  hipDeviceReset();
  return 0;
}
